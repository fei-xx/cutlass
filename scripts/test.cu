#include "hip/hip_runtime.h"


#include <iostream>
#include <memory>
#include <random>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"

#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"

#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"



// We compile all models with -fvisibility=hidden. Any symbols that need to be
// exposed in the final shared library must be declared with PT_EXPORT to make
// them visible.
#ifdef __GNUC__ // Applies to any compiler with GNU extensions (clang and g++)
#define PT_EXPORT __attribute__((__visibility__("default")))
#else
#ifdef _WIN32
#define PT_EXPORT __declspec(dllexport)
#else
#define PT_EXPORT
#endif
#endif

using bfloat16 = hip_bfloat16;

#define CUTLASS_CHECK(status)                                                      \
{                                                                                  \
  cutlass::Status error = status;                                                  \
  if (error != cutlass::Status::kSuccess) {                                        \
    auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +             \
        cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);        \
    std::cerr << msg << std::endl;                                                 \
    throw std::runtime_error(msg);                                                 \
  }                                                                                \
}




using cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_epilogue =
  typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    cute::Shape<cute::_128, cute::_128, cute::_64>,
    cute::Shape<cute::_2,cute::_1,cute::_1>,
    cutlass::epilogue::collective::EpilogueTileAuto,
    cutlass::half_t, cutlass::half_t,
    void, cutlass::layout::RowMajor, 8,
    cutlass::half_t, cutlass::layout::RowMajor, 8,
    cutlass::epilogue::collective::EpilogueScheduleAuto
  >::CollectiveOp;

using cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_mainloop =
  typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    cutlass::half_t, cutlass::layout::RowMajor, 8,
    cutlass::half_t, cutlass::layout::ColumnMajor, 8,
    cutlass::half_t,
    cute::Shape<cute::_128, cute::_128, cute::_64>,
    cute::Shape<cute::_2,cute::_1,cute::_1>,
    cutlass::gemm::collective::StageCountAutoCarveout<
      sizeof(typename cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_epilogue::SharedStorage)>,
  cutlass::gemm::collective::KernelScheduleAuto
  >::CollectiveOp;

// Gemm operator cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8
using cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_base = cutlass::gemm::kernel::GemmUniversal<
    cute::Shape<int,int,int,int>,
    cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_mainloop,
    cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_epilogue>;

// Define named type
struct cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8 :
  public cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_base { };


  using cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_device_type = cutlass::gemm::device::GemmUniversalAdapter<cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8>;


// When workspace_size is not a nullptr, populates requested workspace_size and returns.
// Otherwise, compuates the Gemm kernel using the given workspace ptr.
extern "C" {
PT_EXPORT int cuda_cutlass_gemm(const half* X, const half* W, half* Y, size_t* workspace_size, uint8_t* workspace, hipStream_t stream) {
  try {
  printf("X: %p\n", (void*)(X));
  printf("W: %p\n", (void*)(W));
  // printf("Bias: %p\n", (void*)(Bias));
  printf("Y: %p\n", (void*)(Y));
  printf("workspace size: %p\n", (void*)(workspace_size));
  printf("workspace: %p\n", (void*)(workspace));
  printf("stream: %p\n", (void*)(stream));

  
  {
    if (!X) {
      int64_t X_size = 16000L;
      if (X_size > 0) {
        throw std::runtime_error("input X is null!");
      }
    }
  }

  
  {
    if (!W) {
      int64_t W_size = 16000L;
      if (W_size > 0) {
        throw std::runtime_error("input W is null!");
      }
    }
  }

  
  
  {
    if (!Y) {
      int64_t Y_size = 1000000L;
      if (Y_size > 0) {
        throw std::runtime_error("input Y is null!");
      }
    }
  }


  int64_t B = 1;
  int64_t M = 1000L;
  int64_t K = 16L;
  int64_t N = 1000L;

  using ElementComputeEpilogue = cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_device_type::ElementAccumulator;
  using coord_t = cutlass::gemm::GemmCoord::Index;
  cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_device_type::Arguments arguments;
  
  // Initialize GemmUniversal3xInstance arguments.
  arguments = {
    cutlass::gemm::GemmUniversalMode::kGemm,  // GemmUniversalMode mode
    {
      static_cast<coord_t>(M),
      static_cast<coord_t>(N),
      static_cast<coord_t>(K),
      static_cast<coord_t>(B)
    }, // ProblemShape problem_shape
    {
      (cutlass::half_t*)(X),  // ElementA const* ptr_A
      { 16L /* stride_x0 */, cute::Int<1>{} /* stride_x1 */, 0 /* batch_stride_x */},  // StrideA dA
      (cutlass::half_t*)(W),  // ElementB const* ptr_B
      { 16L /* stride_w1 */, cute::Int<1>{} /* stride_w0 */, 0 /* batch_stride_w */},  // StrideB dB
    },  // MainloopArguments mainloop
    
    {
      {ElementComputeEpilogue(1), ElementComputeEpilogue(0)},  // typename ThreadEpilogueOp::Params thread
      nullptr,  // ElementC const* ptr_C
      { cute::Int<1>{} /* stride_bias0 */, cute::Int<1>{} /* stride_bias1 */, 0 /* batch_stride_bias */},  // StrideC dC
      (cutlass::half_t*)(Y),  // ElementD const* ptr_D
      { 1000L /* stride_y0 */, cute::Int<1>{} /* stride_y1 */, 0 /* batch_stride_y */},  // StrideD dD
    },  // EpilogueArguments epilogue, no TMA
  };

  cutlass3x_sm90_tensorop_h64x128x16gemm_f16_f16_f16_f16_f16_128x128x64_2x1x1_0_tnt_align8_device_type gemm_op;

  if (workspace_size) {
    *workspace_size = gemm_op.get_workspace_size(arguments);
    return 0;
  }

  {
    auto status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
  }
  {
    auto status = gemm_op.initialize(arguments, workspace, stream);
    CUTLASS_CHECK(status);
  }
  {
    auto status = gemm_op(stream);
    CUTLASS_CHECK(status);
  }
  }
  catch (...) {
    return -1;
  }

  return 0;
}
}